
#include <hip/hip_runtime.h>
#include <stdio.h>

// kernel function
__global__ void hello_world_kernel() {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello from thread %d\n", threadId);
}

int main() {
    hello_world_kernel <<<2, 4>>> ();
    hipDeviceSynchronize();
    return 0;
}